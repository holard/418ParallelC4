#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "LocklessMap.h"
#include "board.h"

__device__ int 
compute_thing_kernel(int input) {
    int acc = 0;
    for (int i = 0; i < 1000; i++) {
        acc += i % (input+1);
    }
    return acc;
}

__device__ int
get(Board& b, int row, int col) {
    if (row < 0 || row >= ROWS || col < 0 || col >= COLS) {
        return 0;
    }
    return b.state[row][col];
}

__device__ int
score_board(Board& b) {
    int sum = 0;
    compute_thing_kernel(50);
    for (int row = 0; row < ROWS; row++) {
        for (int col = 0; col < COLS; col++) {
            int p = b.state[row][col];
            if (p == 0)
                continue;
            int length = 1;
            while (get(b, row,col+length) == p) {
                length += 1;
            }
            if (length >= 4)
                return p*INF;
            sum += p*length;
            length = 1;
            while (get(b, row+length, col) == p) {
                length += 1;
            }
            if (length >= 4)
                return p*INF;
            sum += p*length;
            length = 1;
            while (get(b, row+length, col+length) == p) {
                length += 1;
            }
            if (length >= 4)
                return p*INF;
            sum += p*length;
            length = 1;
            while (get(b, row+length, col-length) == p) {
                length += 1;
            }
            if (length >= 4)
                return p*INF;
            sum += p*length;
        }
    }
    return sum;
}

__global__ void
score_kernel(Board* device_array, int count, int* result_array) {
    int i = blockIdx.x * 256 + threadIdx.x;
    if (i >= count)
        return;

    int s = score_board(device_array[i]);
    result_array[i] = s;
}

void score_base_cuda (Frontier& input, int player, LocklessMap& result) {
    int count = input.count;
    Board* device_array;
    int* result_array;
    hipMalloc(&device_array, sizeof(Board) * count);
    hipMalloc(&result_array, sizeof(int) * count);
    hipMemcpy(device_array, input.buffer, sizeof(Board) * count, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int numBlocks = (count+threadsPerBlock-1)/threadsPerBlock;
    score_kernel<<<numBlocks, threadsPerBlock>>>(device_array, count, result_array);
    hipDeviceSynchronize();

    int* host_result = new int[count];
    hipMemcpy(host_result, result_array, sizeof(int) * count, hipMemcpyDeviceToHost);

    for (int i = 0; i < count; i++) {
        Board& b = input.buffer[i];
        Key k = b.getKey();
        result.put_seq(k, host_result[i]);
    }

    hipFree(device_array);
    hipFree(result_array);
    delete [] host_result;
    return;
}


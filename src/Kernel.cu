#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

__global__ void
bar(int* array) {
    int n = threadIdx.x;
    array[n] = n + 10;
}

int kernel() {

    int* device_x;
    int result;

    hipMalloc(&device_x, sizeof(int) * 4);
    bar<<<1,4>>>(device_x);
    hipDeviceSynchronize();

    hipMemcpy(&result, device_x+3, sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(device_x);
    return result;
}

